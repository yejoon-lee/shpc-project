#include "hip/hip_runtime.h"
// Invalid but no-error CUDA code

#include "layer.h"

#include <hip/hip_runtime.h>
#include <mpi.h>
#include <omp.h>

#define DIV_CEIL(a, b) (((a) + (b)-1) / (b))

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)


// CUDA Kernel for token_pos_embedding
__global__ void token_pos_embedding_kernel(int *in, float *wte, float *wpe, float *out, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < s && j < H) {
        // out[b,i,j] = wte[in[b,i],j] + wpe[i,j]
        out[(b * s * H) + i * H + j] = wte[in[b * s + i] * H + j] + wpe[i * H + j];
    }
}

/* Token + Positional Embedding
 * @param [in1]  in: [B, s]
 * @param [in2] wte: [NUM_VOCAB, H]
 * @param [in3] wpe: [MAX_SEQ_LEN, H]
 * @param [out] out: [B, s, H]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
void token_pos_embedding(vector<int> *in, Tensor *wte, Tensor *wpe,
                              Tensor *out, int prompt_size) {
  size_t s = prompt_size;
  size_t B = in->size();
  size_t H = wte->shape[1];

  // Concatenate the input vectors into a single array
  std::vector<int> concatenated_input;
  concatenated_input.reserve(B * s);
  for (size_t i = 0; i < B; ++i) {
      concatenated_input.insert(concatenated_input.end(), in[i].begin(), in[i].end());
  }

  // `in` is on the host, so we need to copy it to the device
  int *d_in;
  CHECK_CUDA(hipMalloc(&d_in, B*s * sizeof(int)));
  CHECK_CUDA(hipMemcpy(d_in, concatenated_input.data(), B*s * sizeof(int), hipMemcpyHostToDevice));

  dim3 blockDim(16, 1, 16);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H, blockDim.z));

  token_pos_embedding_kernel<<<gridDim, blockDim>>>(d_in, wte->buf, wpe->buf, out->buf, B, s, H);

  CHECK_CUDA(hipFree(d_in));
}

/* GELU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
// CUDA Kernel for GELU
__global__ void gelu_kernel(float *inout, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = inout[idx];
        inout[idx] = 0.5 * x * (1.f + tanh(sqrt(2.f / MATH_PI) * (x + 0.044715f * x * x * x)));
    }
}

// GELU using CUDA
void gelu(Tensor *inout) {
  size_t N = inout->num_elem();

  gelu_kernel<<<DIV_CEIL(N, 256), 256>>>(inout->buf, N);
  CHECK_CUDA(hipGetLastError());
}


// CUDA Kernel for softmax
__global__ void softmax_kernel(float *inout, size_t B, size_t s, size_t H) {
    // Calculate the thread indices
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && i < s){
      // Find the maximum value in the row
      float max_val = inout[b * s * H + i * H];
      for (size_t j = 1; j < H; j++) {
          if (inout[b * s * H + i * H + j] > max_val) {
              max_val = inout[b * s * H + i * H + j];
          }
      }

      // Compute the denominator
      float sum = 0.0;
      for (size_t j = 0; j < H; j++) {
        inout[b * s * H + i * H + j] = exp(inout[b * s * H + i * H + j] - max_val);
        sum += inout[b * s * H + i * H + j];
      }

      // Normalize the row
      for (size_t j = 0; j < H; j++) {
        inout[b * s * H + i * H + j] /= sum;
      }
    }
}

/* Softmax (w/ Max Trick)
 * @param [in & out] inout: [B, s, H]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
void softmax(Tensor *inout) {
    size_t B = inout->shape[0];
    size_t s = inout->shape[1];
    size_t H = inout->shape[2];  // actually equal to s (used on attention scores)

    // Define grid and block dimensions
    dim3 blockDim(32, 8);
    dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y));

    // Launch the kernel
    softmax_kernel<<<gridDim, blockDim>>>(inout->buf, B, s, H);
    CHECK_CUDA(hipGetLastError());
}



// CUDA Kernel for layer normalization
__global__ void layer_norm_kernel(float *inout, float *gamma, float *beta, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && i < s){
        float eps = 1e-5;
        float mean = 0;
        float var = 0;

        // Compute the mean and variance
        for (size_t j = 0; j < H; j++) {
            mean += inout[b * s * H + i * H + j];
            var += inout[b * s * H + i * H + j] * inout[b * s * H + i * H + j];
        }
        mean /= H;
        var = var / H - mean * mean;

        // Normalize the row
        for (size_t j = 0; j < H; j++) {
            inout[b * s * H + i * H + j] = (inout[b * s * H + i * H + j] - mean) *
            (1.0 / sqrt(var + eps)) * gamma[j] + beta[j];
        }
    }
}

/* Layer Normalization
 * @param [in1 & out] inout: [B, s, H]
 * @param [in2]       gamma: [H]
 * @param [in3]        beta: [H]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
void layer_norm(Tensor *inout, Tensor *gamma, Tensor *beta) {
  size_t B = inout->shape[0];
  size_t s = inout->shape[1];
  size_t H = inout->shape[2];

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y));

    // Launch the kernel
    layer_norm_kernel<<<gridDim, blockDim>>>(inout->buf, gamma->buf, beta->buf, B, s, H);
    CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for linear
__global__ void linear_kernel(float *in, float *W, float *Bias, float *out, size_t B, size_t M, size_t K, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        float sum = 0.0;
        for (size_t k = 0; k < K; k++) {
            sum += in[b * M * K + i * K + k] * W[k * N + j];
        }
        out[b * M * N + i * N + j] = sum + Bias[j];
    }
}

/* Linear
 * @param [in1]  in: [B, M, K]
 * @param [in2]   w: [K, N]
 * @param [in3]   b: [N]
 * @param [out] out: [B, M, N]
 */
void linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t B = in->shape[0];
  size_t M = in->shape[1];
  size_t K = in->shape[2];
  size_t N = w->shape[1];

  // Define grid and block dimensions
  dim3 blockDim(8, 8, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  linear_kernel<<<gridDim, blockDim>>>(in->buf, w->buf, b->buf, out->buf, B, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for matmul_attnscore
__global__ void matmul_attnscore_kernel(float *in1, float *in2, float *out, size_t B, size_t M, size_t K, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            sum += in1[b * M * K + i * K + k] * in2[b * K * N + k * N + j];
        }
        out[b * M * N + i * N + j] = sum;
    }
}

/* Matmul(Attention Score)
 * @param [in1]  in1: [B, M, K]
 * @param [in2]  in2: [B, K, N]
 * @param [out]  out: [B, M, N]
 */
void matmul_attnscore(Tensor *in1, Tensor *in2, Tensor *out) {
  size_t B = in1->shape[0];
  size_t M = in1->shape[1]; // s
  size_t K = in1->shape[2]; // H_
  size_t N = in2->shape[2]; // s

  // Define grid and block dimensions
  dim3 blockDim(64, 2, 2);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  matmul_attnscore_kernel<<<gridDim, blockDim>>>(in1->buf, in2->buf, out->buf, B, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for matmul_attnout
__global__ void matmul_attnout_kernel(float *in1, float *in2, float *out, size_t B, size_t M, size_t K, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            sum += in1[b * M * K + i * K + k] * in2[b * K * N + k * N + j];
        }
        out[b * M * N + i * N + j] = sum;
    }
}

/* Matmul(Attention Output)
 * @param [in1]  in1: [B, M, K]
 * @param [in2]  in2: [B, K, N]
 * @param [out]  out: [B, M, N]
 */
void matmul_attnout(Tensor *in1, Tensor *in2, Tensor *out) {
  size_t B = in1->shape[0];
  size_t M = in1->shape[1]; // s
  size_t K = in1->shape[2]; // s
  size_t N = in2->shape[2]; // H_

  // Define grid and block dimensions
  dim3 blockDim(64, 2, 2);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  matmul_attnout_kernel<<<gridDim, blockDim>>>(in1->buf, in2->buf, out->buf, B, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for matmul_ffn
__global__ void matmul_ffn_kernel(float *in1, float *in2, float *out, size_t B, size_t M, size_t K, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            // out[b, i, j] = in1[b, i, k] * in2[k, j]
            sum += in1[b * M * K + i * K + k] * in2[k * N + j];
        }
        out[b * M * N + i * N + j] = sum;
    }
}

/* Matmul(FFN)
 * @param [in1]  in1: [B, M, K]
 * @param [in2]  in2: [K, N]
 * @param [out]  out: [B, M, N]
 */
void matmul_ffn(Tensor *in1, Tensor *in2, Tensor *out) {
  size_t B = in1->shape[0];
  size_t M = in1->shape[1]; // s
  size_t K = in1->shape[2]; // H
  size_t N = in2->shape[1]; // V

  // Define grid and block dimensions
  dim3 blockDim(16, 1, 16);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  matmul_ffn_kernel<<<gridDim, blockDim>>>(in1->buf, in2->buf, out->buf, B, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for transpose
__global__ void transpose_kernel(float *in, float *out, size_t M, size_t N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N) {
        out[j * M + i] = in[i * N + j];
    }
}

/* Transpose_batch
 * @param [in1]  in: [M, N]
 * @param [out] out: [N, M]
*/
void transpose(Tensor *in, Tensor *out) {
  size_t M = in->shape[0]; // V
  size_t N = in->shape[1]; // H

  // Define grid and block dimensions
  dim3 blockDim(32, 8);
  dim3 gridDim(DIV_CEIL(M, blockDim.x), DIV_CEIL(N, blockDim.y));

  // Launch the kernel
  transpose_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, M, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for transpose_batch
__global__ void transpose_batch_kernel(float *in, float *out, size_t B, size_t M, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        out[b * N * M + j * M + i] = in[b * M * N + i * N + j];
    }
}

/* Transpose_batch
 * @param [in1]  in: [B, M, N]
 * @param [out] out: [B, N, M]
*/
void transpose_batch(Tensor *in, Tensor *out) {
  size_t B = in->shape[0];
  size_t M = in->shape[1];
  size_t N = in->shape[2];

  // Define grid and block dimensions
  dim3 blockDim(16, 4, 4);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  transpose_batch_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, M, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for scaling
__global__ void scaling_kernel(float *inout, float scale, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { inout[idx] *= scale; }
}

/* Scaling
 * @param [in1 & out] inout: [N]
 * @param [in2]       scale: [1]
 * 'N' is the number of elements in the tensor.
 */
void scaling(Tensor *inout, float scale) {
  size_t N = inout->num_elem();

  scaling_kernel<<<DIV_CEIL(N, 256), 256>>>(inout->buf, scale, N);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for generate mask
__global__ void generate_mask_kernel(float *inout, size_t s) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < s && j < s) {
        inout[i * s + j] = (i >= j) ? 0 : -1e10;
    }
}

/* Generate mask
 * @param [in & out] inout: [s, s]
 * 's' is the number of tokens in the prompt.
 */
void generate_mask(Tensor *inout) {
  size_t s = inout->shape[0];

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(s, blockDim.x), DIV_CEIL(s, blockDim.y));

  // Launch the kernel
  generate_mask_kernel<<<gridDim, blockDim>>>(inout->buf, s);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for copy
__global__ void copy_kernel(float *in, float *out, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { out[idx] = in[idx]; }
}

/* Copy
 * @param [in1]  in: [N]
 * @param [out] out: [N]
 * 'N' is the number of elements in the tensor.
 */
void copy(Tensor *in, Tensor *out) {
  size_t N = in->num_elem();

  copy_kernel<<<DIV_CEIL(N, 256), 256>>>(in->buf, out->buf, N);
  CHECK_CUDA(hipGetLastError());
}


// CUDA Kernel for add
__global__ void add_kernel(float *inout, float *x, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) { inout[idx] += x[idx]; }
}

/* Add using CUDA GPU
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
void add(Tensor *inout, Tensor *x) {
  size_t N = inout->num_elem();

  add_kernel<<<(N + 255) / 256, 256>>>(inout->buf, x->buf, N);
  CHECK_CUDA(hipGetLastError());
}


__global__ void add_batch_kernel(float *inout, float *x, size_t B, size_t N) {
  size_t b = blockIdx.x * blockDim.x + threadIdx.x;
  size_t i = blockIdx.y * blockDim.y + threadIdx.y;

  if (b < B && i < N) { 
    inout[b * N + i] += x[i];
    }
}

/* Add using CUDA GPU
 * @param [in1 & out] inout: [B, M, N]
 * @param [in2]           x: [M, N]
 * 'B' is the batch size.
 * 'N' is the number of elements in the tensor.
 */
void add_batch(Tensor *inout, Tensor *x) {
  size_t B = inout->shape[0];
  size_t N = inout->num_elem() / B;

  // Treat M*N as a single dimension
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(N, blockDim.y));

  add_batch_kernel<<<gridDim, blockDim>>>(inout->buf, x->buf, B, N);
  CHECK_CUDA(hipGetLastError());
}

__global__ void split_qkv_kernel(float *in, float *out, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && j < s && k < H / 3) {
      for (size_t i = 0; i < 3; i++) {
        // out[b, i, j, k] = in[b, j, i * (H / 3) + k]
        out[(b * s * H) + i * s * (H / 3) + j * (H / 3) + k] = in[(b * s * H) + j * H + i * (H / 3) + k];
      }
    }
}

/* Split into QKV
 * @param [in1]  in: [B, s, H]
 * @param [out] out: [B, 3, s, H/3]
 */
void split_qkv(Tensor *in, Tensor *out) {
  size_t B = in->shape[0];
  size_t s = in->shape[1];
  size_t H = in->shape[2];

  // Define grid and block dimensions
  dim3 blockDim(16, 2, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H / 3, blockDim.z));

  // Launch the kernel
  split_qkv_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, s, H);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for split_head
__global__ void split_head_kernel(float *in, float *out, size_t B, size_t s, size_t H, size_t n_head) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && j < n_head && k < s) {
      for (size_t i = 0; i < 3; i++) {
        for (size_t l = 0; l < H / n_head; l++) {
            // out[b, i, j, k, l] = in[b, i, k, j * (H / n_head) + l]
            out[(b * 3 * s * H) + i * s * H + j * s * H / n_head + k * H / n_head + l] =
                in[(b * 3 * s * H) + i * s * H + k * H + j * H / n_head + l];
        }
      }
    }
}

/* Split into heads
 * @param [in1]  in: [B, 3, s, H]
 * @param [out] out: [B, 3, n_head, s, H/n_head]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 * 'n_head' is the number of heads.
 */
void split_head(Tensor *in, size_t n_head, Tensor *out) {
  size_t B = in->shape[0];
  size_t s = in->shape[2];
  size_t H = in->shape[3];

  // Define grid and block dimensions
  dim3 blockDim(16, 2, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), n_head);

  // Launch the kernel
  split_head_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, s, H, n_head);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for extract_qkv
__global__ void extract_qkv_kernel(float *in, size_t head_idx, size_t n_head, float *q, float *k, float *v, size_t B, size_t s, size_t H_) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (b < B && i < s && j < H_) {
        // q[b, i, j] = in[b, 0, head_idx, i, j]
        q[(b * s * H_) + i * H_ + j] = in[(b * 3 * n_head * s * H_) + 0 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
        k[(b * s * H_) + i * H_ + j] = in[(b * 3 * n_head * s * H_) + 1 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
        v[(b * s * H_) + i * H_ + j] = in[(b * 3 * n_head * s * H_) + 2 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
    }   
}

/* Extract Q, K, V from QKV head
 * @param [in1]       in: [B, 3, n_head, s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]        q: [B, s, H_]
 * @param [out]        k: [B, s, H_]
 * @param [out]        v: [B, s, H_]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
void extract_qkv(Tensor *in, size_t head_idx, size_t n_head, Tensor *q, Tensor *k, Tensor *v) {
  size_t B = in->shape[0];
  size_t s = in->shape[3];
  size_t H_ = in->shape[4];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(32, 2, 4);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H_, blockDim.z));

  // Launch the kernel
  extract_qkv_kernel<<<gridDim, blockDim>>>(in->buf, head_idx, n_head, q->buf, k->buf, v->buf, B, s, H_);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for merge_head
__global__ void merge_head_kernel(float *in, size_t head_idx, size_t n_head, float *out, size_t B, size_t s, size_t H_) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < s && j < H_) {
        out[b * n_head * s * H_ + head_idx * s * H_ + i * H_ + j] = in[b * s * H_ + i * H_ + j];
    }
}

/* Merge each heads
 * @param [in1]       in: [B, s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]      out: [B, n_head, s, H_]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
void merge_head(Tensor *in, size_t head_idx, size_t n_head, Tensor *out) {
  size_t B = in->shape[0];
  size_t s = in->shape[1];
  size_t H_ = in->shape[2];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(32, 2, 4);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H_, blockDim.z));

  // Launch the kernel
  merge_head_kernel<<<gridDim, blockDim>>>(in->buf, head_idx, n_head, out->buf, B, s, H_);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for concat_head
__global__ void concat_head_kernel(float *in, float *out, size_t B, size_t n_head, size_t s, size_t H_) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < s && j < n_head) {
        for (size_t k = 0; k < H_; k++) {
            out[b * s * (H_ * n_head) + i * (H_ * n_head) + (j * H_ + k)] = in[b * n_head * s * H_ + j * s * H_ + i * H_ + k];
        }
    }
}

/* Concatenate each heads
 * @param [in1]     in: [B, n_head, s, H_]
 * @param [out]    out: [B, s, H]
 * H = H_ * n_head
 * 'B' is the batch size.
 * 'n_head' is the number of heads.
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 */
void concat_head(Tensor *in, Tensor *out) {
  size_t B = in->shape[0];
  size_t n_head = in->shape[1];
  size_t s = in->shape[2];
  size_t H_ = in->shape[3];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(64, 2, 2);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(n_head, blockDim.z));

  // Launch the kernel
  concat_head_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, n_head, s, H_);
  CHECK_CUDA(hipGetLastError());
}

__global__ void top1_sampling_kernel(float *in, int *next_token_ids, size_t B, size_t s, size_t V) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;

    if (b < B) {
        float max = -INFINITY;
        int idx = 0;
        for (size_t i = 0; i < V; i++) {
            if (in[b * s * V + (s - 1) * V + i] > max) {
                max = in[b * s * V + (s - 1) * V + i];
                idx = i;
            }
        }
        next_token_ids[b] = idx;
    }
}

/* Greedy Max Sampling
 * @param  [in1]  in: [B, s, V]
 * @param [out] next_token_ids: [B]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'V' is the number of vocabulary.
 * Device -> Host
 */
void top1_sampling(Tensor *in, int *next_token_ids) {
  size_t B = in->shape[0];
  size_t s = in->shape[1];
  size_t V = in->shape[2];

  // hipMalloc next_token_ids
  int *next_token_ids_d;
  CHECK_CUDA(hipMalloc(&next_token_ids_d, B * sizeof(int)));

  // Define grid and block dimensions
  dim3 blockDim(64);
  dim3 gridDim(DIV_CEIL(B, blockDim.x));

  // Launch the kernel
  top1_sampling_kernel<<<gridDim, blockDim>>>(in->buf, next_token_ids_d, B, s, V);

  // Copy the result back to the host
  CHECK_CUDA(hipMemcpy(next_token_ids, next_token_ids_d, B * sizeof(int), hipMemcpyDeviceToHost));

  // Free the device memory
  CHECK_CUDA(hipFree(next_token_ids_d));
}

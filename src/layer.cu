#include "hip/hip_runtime.h"
// Invalid but no-error CUDA code

#include "layer.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define DIV_CEIL(a, b) (((a) + (b)-1) / (b))

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)


// CUDA Kernel for token_pos_embedding
__global__ void token_pos_embedding_kernel(int *in, float *wte, float *wpe, float *out, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < s && j < H) {
        // out[b,i,j] = wte[in[b,i],j] + wpe[i,j]
        out[(b * s * H) + i * H + j] = wte[in[b * s + i] * H + j] + wpe[i * H + j];
    }
}

/* Token + Positional Embedding
 * @param [in1]  in: [B, s]
 * @param [in2] wte: [NUM_VOCAB, H]
 * @param [in3] wpe: [MAX_SEQ_LEN, H]
 * @param [out] out: [B, s, H]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
void token_pos_embedding(vector<int> in, Tensor *wte, Tensor *wpe,
                              Tensor *out, int prompt_size) {
  size_t s = prompt_size;
  size_t B = in.size() / s;
  size_t H = wte->shape[1];

  // `in` is on the host, so we need to copy it to the device
  int *d_in;
  CHECK_CUDA(hipMalloc(&d_in, B*s * sizeof(int)));
  CHECK_CUDA(hipMemcpy(d_in, in.data(), B*s * sizeof(int), hipMemcpyHostToDevice));

  dim3 blockDim(16, 2, 16);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H, blockDim.z));

  token_pos_embedding_kernel<<<gridDim, blockDim>>>(d_in, wte->buf, wpe->buf, out->buf, B, s, H);

  CHECK_CUDA(hipFree(d_in));
}

/* GELU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
// CUDA Kernel for GELU
__global__ void gelu_kernel(float *inout, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = inout[idx];
        inout[idx] = 0.5 * x * (1.f + tanh(sqrt(2.f / MATH_PI) * (x + 0.044715f * x * x * x)));
    }
}

// GELU using CUDA
void gelu(Tensor *inout) {
  size_t N = inout->num_elem();

  gelu_kernel<<<DIV_CEIL(N, 256), 256>>>(inout->buf, N);
  CHECK_CUDA(hipGetLastError());
}


/* Softmax (w/ Max Trick)
 * @param [in & out] inout: [s, H]
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
__global__ void softmax_kernel(float *inout, size_t s, size_t H) {
    // Calculate the thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < s){
      // Find the maximum value in the row
      float max_val = inout[i * H];
      for (size_t j = 1; j < H; j++) {
          if (inout[i * H + j] > max_val) {
              max_val = inout[i * H + j];
          }
      }

      // Compute the denominator
      float sum = 0.0;
      for (size_t j = 0; j < H; j++) {
        inout[i * H + j] = exp(inout[i * H + j] - max_val);
        sum += inout[i * H + j];
      }

      // Normalize the row
      for (size_t j = 0; j < H; j++) {
        inout[i * H + j] /= sum;
      }
    }
}

void softmax(Tensor *inout) {
    size_t s = inout->shape[0];
    size_t H = inout->shape[1];

    // Define grid and block dimensions
    dim3 blockDim(32); // warp = 32 threads
    dim3 gridDim(DIV_CEIL(s, blockDim.x));

    // Launch the kernel
    softmax_kernel<<<gridDim, blockDim>>>(inout->buf, s, H);
    CHECK_CUDA(hipGetLastError());
}



// CUDA Kernel for layer normalization
__global__ void layer_norm_kernel(float *inout, float *gamma, float *beta, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && i < s){
        float eps = 1e-5;
        float mean = 0;
        float var = 0;

        // Compute the mean and variance
        for (size_t j = 0; j < H; j++) {
            mean += inout[b * s * H + i * H + j];
            var += inout[b * s * H + i * H + j] * inout[b * s * H + i * H + j];
        }
        mean /= H;
        var = var / H - mean * mean;

        // Normalize the row
        for (size_t j = 0; j < H; j++) {
            inout[b * s * H + i * H + j] = (inout[b * s * H + i * H + j] - mean) *
            (1.0 / sqrt(var + eps)) * gamma[j] + beta[j];
        }
    }
}

/* Layer Normalization
 * @param [in1 & out] inout: [B, s, H]
 * @param [in2]       gamma: [H]
 * @param [in3]        beta: [H]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 */
void layer_norm(Tensor *inout, Tensor *gamma, Tensor *beta) {
  size_t B = inout->shape[0];
  size_t s = inout->shape[1];
  size_t H = inout->shape[2];

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y));

    // Launch the kernel
    layer_norm_kernel<<<gridDim, blockDim>>>(inout->buf, gamma->buf, beta->buf, B, s, H);
    CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for linear
__global__ void linear_kernel(float *in, float *W, float *Bias, float *out, size_t B, size_t M, size_t K, size_t N) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && i < M && j < N) {
        float sum = 0.0;
        for (size_t k = 0; k < K; k++) {
            sum += in[b * M * K + i * K + k] * W[k * N + j];
        }
        out[b * M * N + i * N + j] = sum + Bias[j];
    }
}

/* Linear
 * @param [in1]  in: [B, M, K]
 * @param [in2]   w: [K, N]
 * @param [in3]   b: [N]
 * @param [out] out: [B, M, N]
 */
void linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t B = in->shape[0];
  size_t M = in->shape[1];
  size_t K = in->shape[2];
  size_t N = w->shape[1];

  // Define grid and block dimensions
  dim3 blockDim(8, 8, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(M, blockDim.y), DIV_CEIL(N, blockDim.z));

  // Launch the kernel
  linear_kernel<<<gridDim, blockDim>>>(in->buf, w->buf, b->buf, out->buf, B, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

/* Matmul
 * @param [in1]  in1: [M, K]
 * @param [in2]  in2: [K, N]
 * @param [out]  out: [M, N]
 */
// CUDA Kernel for matmul
__global__ void matmul_kernel(float *in1, float *in2, float *out, size_t M, size_t K, size_t N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            sum += in1[i * K + k] * in2[k * N + j];
        }
        out[i * N + j] = sum;
    }
}

// Matmul using CUDA
void matmul(Tensor *in1, Tensor *in2, Tensor *out) {
  size_t M = in1->shape[0];
  size_t K = in1->shape[1];
  size_t N = in2->shape[1];

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(N, blockDim.x), DIV_CEIL(M, blockDim.y));

  // Launch the kernel
  matmul_kernel<<<gridDim, blockDim>>>(in1->buf, in2->buf, out->buf, M, K, N);
  CHECK_CUDA(hipGetLastError());
}

/* Transpose
 * @param [in1]  in: [M, N]
 * @param [out] out: [N, M]
 */
// CUDA Kernel for transpose
__global__ void transpose_kernel(float *in, float *out, size_t M, size_t N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < M && j < N) {
        out[j * M + i] = in[i * N + j];
    }
}

// Transpose using CUDA
void transpose(Tensor *in, Tensor *out) {
  size_t M = in->shape[0];
  size_t N = in->shape[1];

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(N, blockDim.x), DIV_CEIL(M, blockDim.y));

  // Launch the kernel
  transpose_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, M, N);
  CHECK_CUDA(hipGetLastError());
}

/* Scaling
 * @param [in1 & out] inout: [N]
 * @param [in2]       scale: [1]
 * 'N' is the number of elements in the tensor.
 */
// CUDA Kernel for scaling
__global__ void scaling_kernel(float *inout, float scale, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { inout[idx] *= scale; }
}

// Scaling using CUDA
void scaling(Tensor *inout, float scale) {
  size_t N = inout->num_elem();

  scaling_kernel<<<DIV_CEIL(N, 256), 256>>>(inout->buf, scale, N);
  CHECK_CUDA(hipGetLastError());
}

/* Generate mask
 * @param [in & out] inout: [s, s]
 * 's' is the number of tokens in the prompt.
 */
// CUDA Kernel for generate mask
__global__ void generate_mask_kernel(float *inout, size_t s) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < s && j < s) {
        inout[i * s + j] = (i >= j) ? 0 : -1e10;
    }
}

// Generate mask using CUDA
void generate_mask(Tensor *inout) {
  size_t s = inout->shape[0];

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(s, blockDim.x), DIV_CEIL(s, blockDim.y));

  // Launch the kernel
  generate_mask_kernel<<<gridDim, blockDim>>>(inout->buf, s);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for copy
__global__ void copy_kernel(float *in, float *out, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { out[idx] = in[idx]; }
}

/* Copy
 * @param [in1]  in: [N]
 * @param [out] out: [N]
 * 'N' is the number of elements in the tensor.
 */
void copy(Tensor *in, Tensor *out) {
  size_t N = in->num_elem();

  copy_kernel<<<DIV_CEIL(N, 256), 256>>>(in->buf, out->buf, N);
  CHECK_CUDA(hipGetLastError());
}


/* Add GPU kernel
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
__global__ void add_kernel(float *inout, float *x, size_t N) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) { inout[idx] += x[idx]; }
}

/* Add using CUDA GPU
 * @param [in1 & out] inout: [N]
 * @param [in2]           x: [N]
 * 'N' is the number of elements in the tensor.
 */
void add(Tensor *inout, Tensor *x) {
  size_t N = inout->num_elem();

  add_kernel<<<(N + 255) / 256, 256>>>(inout->buf, x->buf, N);
  CHECK_CUDA(hipGetLastError());
}

__global__ void split_qkv_kernel(float *in, float *out, size_t B, size_t s, size_t H) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && j < s && k < H / 3) {
      for (size_t i = 0; i < 3; i++) {
        // out[b, i, j, k] = in[b, j, i * (H / 3) + k]
        out[(b * s * H) + i * s * (H / 3) + j * (H / 3) + k] = in[(b * s * H) + j * H + i * (H / 3) + k];
      }
    }
}

/* Split into QKV
 * @param [in1]  in: [B, s, H]
 * @param [out] out: [B, 3, s, H/3]
 */
void split_qkv(Tensor *in, Tensor *out) {
  size_t B = in->shape[0];
  size_t s = in->shape[1];
  size_t H = in->shape[2];

  // Define grid and block dimensions
  dim3 blockDim(16, 4, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), DIV_CEIL(H / 3, blockDim.z));

  // Launch the kernel
  split_qkv_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, s, H);
  CHECK_CUDA(hipGetLastError());
}

// CUDA Kernel for split_head
__global__ void split_head_kernel(float *in, float *out, size_t B, size_t s, size_t H, size_t n_head) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (b < B && j < n_head && k < s) {
      for (size_t i = 0; i < 3; i++) {
        for (size_t l = 0; l < H / n_head; l++) {
            // out[b, i, j, k, l] = in[b, i, k, j * (H / n_head) + l]
            out[(b * 3 * s * H) + i * s * H + j * s * H / n_head + k * H / n_head + l] =
                in[(b * 3 * s * H) + i * s * H + k * H + j * H / n_head + l];
        }
      }
    }
}

/* Split into heads
 * @param [in1]  in: [B, 3, s, H]
 * @param [out] out: [B, 3, n_head, s, H/n_head]
 * 'B' is the batch size.
 * 's' is the number of tokens in the prompt.
 * 'H' is the hidden dimension.
 * 'n_head' is the number of heads.
 */
void split_head(Tensor *in, size_t n_head, Tensor *out) {
  size_t B = in->shape[0];
  size_t s = in->shape[2];
  size_t H = in->shape[3];

  // Define grid and block dimensions
  dim3 blockDim(16, 4, 8);
  dim3 gridDim(DIV_CEIL(B, blockDim.x), DIV_CEIL(s, blockDim.y), n_head);

  // Launch the kernel
  split_head_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, B, s, H, n_head);
  CHECK_CUDA(hipGetLastError());
}

/* Extract Q, K, V from QKV head
 * @param [in1]       in: [3, n_head, s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]        q: [s, H_]
 * @param [out]        k: [s, H_]
 * @param [out]        v: [s, H_]
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
// CUDA Kernel for extract_qkv
__global__ void extract_qkv_kernel(float *in, size_t head_idx, size_t n_head, float *q, float *k, float *v, size_t s, size_t H_) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < s && j < H_) {
        q[i * H_ + j] = in[0 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
        k[i * H_ + j] = in[1 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
        v[i * H_ + j] = in[2 * n_head * s * H_ + head_idx * s * H_ + i * H_ + j];
    }   
}

// Extract Q, K, V from QKV head using CUDA
void extract_qkv(Tensor *in, size_t head_idx, size_t n_head, Tensor *q, Tensor *k, Tensor *v) {
  size_t s = in->shape[2];
  size_t H_ = in->shape[3];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(8, 32);
  dim3 gridDim(DIV_CEIL(s, blockDim.x), DIV_CEIL(H_, blockDim.y));

  // Launch the kernel
  extract_qkv_kernel<<<gridDim, blockDim>>>(in->buf, head_idx, n_head, q->buf, k->buf, v->buf, s, H_);
  CHECK_CUDA(hipGetLastError());
}

/* Merge each heads
 * @param [in1]       in: [s, H_]
 * @param [in2] head_idx: [1]
 * @param [in3]   n_head: [1]
 * @param [out]      out: [n_head, s, H_]
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 * 'n_head' is the number of heads.
 */
// CUDA Kernel for merge_head
__global__ void merge_head_kernel(float *in, size_t head_idx, size_t n_head, float *out, size_t s, size_t H_) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < s && j < H_) {
        out[head_idx * s * H_ + i * H_ + j] = in[i * H_ + j];
    }
}

// Merge each heads using CUDA
void merge_head(Tensor *in, size_t head_idx, size_t n_head, Tensor *out) {
  size_t s = in->shape[0];
  size_t H_ = in->shape[1];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(8, 32);
  dim3 gridDim(DIV_CEIL(s, blockDim.x), DIV_CEIL(H_, blockDim.y));

  // Launch the kernel
  merge_head_kernel<<<gridDim, blockDim>>>(in->buf, head_idx, n_head, out->buf, s, H_);
  CHECK_CUDA(hipGetLastError());
}

/* Concatenate each heads
 * @param [in1]     in: [n_head, s, H_]
 * @param [out]    out: [s, H_*n_head]
 * 'n_head' is the number of heads.
 * 's' is the number of tokens in the prompt.
 * 'H_' is the hidden dimension/n_head.
 */
// CUDA Kernel for concat_head
__global__ void concat_head_kernel(float *in, float *out, size_t n_head, size_t s, size_t H_) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < s && j < n_head) {
        for (size_t k = 0; k < H_; k++) {
            out[i * n_head * H_ + j * H_ + k] = in[j * s * H_ + i * H_ + k];
        }
    }
}

// Concatenate each heads using CUDA
void concat_head(Tensor *in, Tensor *out) {
  size_t n_head = in->shape[0];
  size_t s = in->shape[1];
  size_t H_ = in->shape[2];  // = HIDDEN_DIM/NUM_HEAD

  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim(DIV_CEIL(s, blockDim.x), DIV_CEIL(n_head, blockDim.y));

  // Launch the kernel
  concat_head_kernel<<<gridDim, blockDim>>>(in->buf, out->buf, n_head, s, H_);
  CHECK_CUDA(hipGetLastError());
}

/* Greedy Max Sampling
 * @param  [in1]  in: [s, V]
 * @return [ret] out: [1]
 * 's' is the number of tokens in the prompt.
 * 'V' is the number of vocabulary.
 */
int top1_sampling(Tensor *in) {
  size_t s = in->shape[0];
  size_t V = in->shape[1];

  int out = 0;
  float max = -INFINITY;
  for (size_t i = 0; i < V; i++) {
    if (in->buf[(s - 1) * V + i] > max) {
      max = in->buf[(s - 1) * V + i];
      out = i;
    }
  }

  return out;
}
